#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <random>

#define NUM_BINS 4096
#define THREADS_PER_BLOCK_HISTOGRAM 1024
#define THREADS_PER_BLOCK_CONVERT 64
#define INPUT_VALUES_PER_THREAD 8
#define SHARED_MEMORY_BINS_PER_THREAD 4

struct timeval startTime;

void startTimer() {
  // Used "Tutorial: Timing your Kernel - CPU Timer & nvprof" for reference
  gettimeofday(&startTime, NULL);
}

double getTimerSeconds() {
  // Used "Tutorial: Timing your Kernel - CPU Timer & nvprof" for reference
  struct timeval endTime;
  gettimeofday(&endTime, NULL);
  return ((double)endTime.tv_sec + (double)endTime.tv_usec * 1.e-6) - ((double)startTime.tv_sec + (double)startTime.tv_usec * 1.e-6);
}

__global__ void histogram_kernel(unsigned int *input, unsigned int *bins, unsigned int num_elements, unsigned int num_bins) {
  //@@ Insert code below to compute histogram of input using shared memory and atomics
  const int id = blockIdx.x * blockDim.x + threadIdx.x;

  // Shared memory stores bins corresponding to this block
  __shared__ unsigned int binsInShared[NUM_BINS];

  // All bins this thread should initialize to 0
  const int startSharedBin = threadIdx.x * SHARED_MEMORY_BINS_PER_THREAD;
  const int endSharedBin = (threadIdx.x + 1) * SHARED_MEMORY_BINS_PER_THREAD;

  // All input values this thread should process
  const int startInputValue = id * INPUT_VALUES_PER_THREAD;
  const int endInputValue = (id + 1) * INPUT_VALUES_PER_THREAD;

  // Set shared memory to 0
  if (startSharedBin < NUM_BINS) {
    // Do not go outside existing bins
    if (endSharedBin > NUM_BINS) {
      endSharedBin = NUM_BINS;
    }

    // Thread sets its bins to 0
    for (int i = startSharedBin; i < endSharedBin; ++i) {
      binsInShared[i] = 0;
    }
  }

  __syncthreads();

  // Add this threads input values to corresponding bins in shared memory
  if (startInputValue < num_elements) {
    // Do not go outside existing input values
    if (endInputValue > num_elements) {
      endInputValue = num_elements;
    }

    // Thread adds its input values to the shared bins
    for (int i = startInputValue; i < endInputValue; ++i) {
      atomicAdd(&binsInShared[input[i]], 1);
    }
  }

  __syncthreads();

  // Merge bins from the different shared memories (blocks) to global memory
  if (startSharedBin < NUM_BINS) {
    // Thread adds the values of its shared bins to the global bins
    for (int i = startSharedBin; i < endSharedBin; ++i) {
      if (binsInShared[i] != 0) {
        atomicAdd(&bins[i], binsInShared[i]);
      }
    }
  }
}

__global__ void convert_kernel(unsigned int *bins, unsigned int num_bins) {
  //@@ Insert code below to clean up bins that saturate at 127
  const int id = blockIdx.x * blockDim.x + threadIdx.x;

  // Each thread checks one bin
  if (id < num_bins) {
    if (bins[id] > 127) {
      bins[id] = 127;
    }
  }
}


int main(int argc, char **argv) {
  int inputLength;
  unsigned int *hostInput;
  unsigned int *hostBins;
  unsigned int *resultRef;
  unsigned int *deviceInput;
  unsigned int *deviceBins;

  //@@ Insert code below to read in inputLength from args
  if (argc != 2) {
    printf("Incorrect number of arguments.\nUsage: ./lab3_ex3.out <inputLength>\n");
    return -1;
  }

  inputLength = atoi(argv[1]);

  printf("The input length is %d\n", inputLength);

  //@@ Insert code below to allocate Host memory for input and output
  hostInput = (unsigned int*) malloc(inputLength * sizeof(unsigned int));
  hostBins = (unsigned int*) malloc(NUM_BINS * sizeof(unsigned int));
  resultRef = (unsigned int*) calloc(NUM_BINS, NUM_BINS * sizeof(unsigned int));

  //@@ Insert code below to initialize hostInput to random numbers whose values range from 0 to (NUM_BINS - 1)
  srand(time(NULL));

  for (int i = 0; i < inputLength; ++i) {
    hostInput[i] = (unsigned int)(((double)rand() / (double)RAND_MAX) * NUM_BINS);
  }

  //@@ Insert code below to create reference result in CPU
  for (int i = 0; i < inputLength; ++i) {
    int binNum = hostInput[i];

    if (resultRef[binNum] < 127) {
      resultRef[binNum]++;
    }
  }

  //@@ Insert code below to allocate GPU memory here
  hipMalloc(&deviceInput, inputLength * sizeof(unsigned int));
  hipMalloc(&deviceBins, NUM_BINS * sizeof(unsigned int));

  //@@ Insert code to Copy memory to the GPU here
  hipMemcpy(deviceInput, hostInput, inputLength * sizeof(unsigned int), hipMemcpyHostToDevice);

  //@@ Insert code to initialize GPU results
  hipMemset(deviceBins, 0, NUM_BINS * sizeof(unsigned int));

  //@@ Initialize the grid and block dimensions here
  dim3 blockHistogram(THREADS_PER_BLOCK_HISTOGRAM);
  dim3 gridHistogram(((inputLength / INPUT_VALUES_PER_THREAD) + THREADS_PER_BLOCK_HISTOGRAM - 1) / THREADS_PER_BLOCK_HISTOGRAM);

  printf("Threads per Block (histogram_kernel): %i\nBlocks (histogram_kernel): %i\n", THREADS_PER_BLOCK_HISTOGRAM, ((inputLength / INPUT_VALUES_PER_THREAD) + THREADS_PER_BLOCK_HISTOGRAM - 1) / THREADS_PER_BLOCK_HISTOGRAM);

  startTimer();

  //@@ Launch the GPU Kernel here
  histogram_kernel<<<gridHistogram, blockHistogram>>>(deviceInput, deviceBins, inputLength, NUM_BINS);
  hipDeviceSynchronize();

  double timeHistogram = getTimerSeconds();

  //@@ Initialize the second grid and block dimensions here
  dim3 blockConvert(THREADS_PER_BLOCK_CONVERT);
  dim3 gridConvert((NUM_BINS + THREADS_PER_BLOCK_CONVERT - 1) / THREADS_PER_BLOCK_CONVERT);

  printf("Threads per Block (convert_kernel): %i\nBlocks (convert_kernel): %i\n", THREADS_PER_BLOCK_CONVERT, (NUM_BINS + THREADS_PER_BLOCK_CONVERT - 1) / THREADS_PER_BLOCK_CONVERT);

  startTimer();

  //@@ Launch the second GPU Kernel here
  convert_kernel<<<gridConvert, blockConvert>>>(deviceBins, NUM_BINS);
  hipDeviceSynchronize();

  double timeConvert = getTimerSeconds();

  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostBins, deviceBins, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost);

  printf("Bins: ");
  int diffs = 0;

  //@@ Insert code below to compare the output with the reference
  for (int i = 0; i < NUM_BINS; ++i) {
    if (hostBins[i] != resultRef[i]) {
      diffs++;
      printf("Bins %i differentiates, found %i instead of %i\n", i, hostBins[i], resultRef[i]);
    }

    // Print the values of all bins on a single line
    if (i < NUM_BINS - 1) {
      printf("%i, ", hostBins[i]);
    }
    else {
      printf("%i\n", hostBins[i]);
    }
  }

  printf("Number of errors: %i/%i\n", diffs, NUM_BINS);

  //@@ Free the GPU memory here
  hipFree(deviceInput);
  hipFree(deviceBins);

  //@@ Free the CPU memory here
  free(hostInput);
  free(hostBins);
  free(resultRef);

  printf("histogram_kernel: %f\nconvert_kernel: %f\nTotal: %f\n", timeHistogram, timeConvert, timeHistogram+timeConvert);

  printf("Done.\n");

  return 0;
}
