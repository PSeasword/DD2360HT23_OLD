#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>

#define DataType double
#define THREADS_PER_BLOCK 64
#define MAXIMUM_RANDOM_VALUE 1000

struct timeval startTime;

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
  //@@ Insert code to implement vector addition here
  int id = blockIdx.x * blockDim.x + threadIdx.x;

  if (id < len) {
    out[id] = in1[id] + in2[id];
  }
}

//@@ Insert code to implement timer start
void startTimer() {
  // Used "Tutorial: Timing your Kernel - CPU Timer & nvprof" for reference
  gettimeofday(&startTime, NULL);
}

//@@ Insert code to implement timer stop
double getTimerSeconds() {
  // Used "Tutorial: Timing your Kernel - CPU Timer & nvprof" for reference
  struct timeval endTime;
  gettimeofday(&endTime, NULL);
  return ((double)endTime.tv_sec + (double)endTime.tv_usec * 1.e-6) - ((double)startTime.tv_sec + (double)startTime.tv_usec * 1.e-6);
}

int main(int argc, char **argv) {
  int inputLength;
  DataType *hostInput1;
  DataType *hostInput2;
  DataType *hostOutput;
  DataType *resultRef;
  DataType *deviceInput1;
  DataType *deviceInput2;
  DataType *deviceOutput;
  double timeHostToDevice;
  double timeKernel;
  double timeDeviceToHost;

  //@@ Insert code below to read in inputLength from args
  if (argc != 2) {
    printf("Incorrect number of arguments.\nUsage: ./lab3_ex1.out <inputLength>\n");
    return -1;
  }

  inputLength = atoi(argv[1]);

  printf("The input length is %d\n", inputLength);

  //@@ Insert code below to allocate Host memory for input and output
  hostInput1 = (DataType*) malloc(inputLength * sizeof(DataType));
  hostInput2 = (DataType*) malloc(inputLength * sizeof(DataType));
  hostOutput = (DataType*) malloc(inputLength * sizeof(DataType));
  resultRef = (DataType*) malloc(inputLength * sizeof(DataType));

  //@@ Insert code below to initialize hostInput1 and hostInput2 to random numbers, and create reference result in CPU
  srand(time(NULL));

  for (int i = 0; i < inputLength; ++i) {
    hostInput1[i] = ((float)rand() / (float)RAND_MAX) * MAXIMUM_RANDOM_VALUE;
    hostInput2[i] = ((float)rand() / (float)RAND_MAX) * MAXIMUM_RANDOM_VALUE;
    resultRef[i] = hostInput1[i] + hostInput2[i];
  }

  //@@ Insert code below to allocate GPU memory here
  hipMalloc(&deviceInput1, inputLength * sizeof(DataType));
  hipMalloc(&deviceInput2, inputLength * sizeof(DataType));
  hipMalloc(&deviceOutput, inputLength * sizeof(DataType));

  startTimer();

  //@@ Insert code to below to Copy memory to the GPU here
  hipMemcpy(deviceInput1, hostInput1, inputLength * sizeof(DataType), hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2, inputLength * sizeof(DataType), hipMemcpyHostToDevice);

  timeHostToDevice = getTimerSeconds();

  //@@ Initialize the 1D grid and block dimensions here
  dim3 block(THREADS_PER_BLOCK);
  dim3 grid((inputLength + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);

  printf("Threads per block: %d\nNumber of thread blocks: %d\n", block.x, grid.x);

  startTimer();

  //@@ Launch the GPU Kernel here
  vecAdd<<<grid, block>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);
  hipDeviceSynchronize();

  timeKernel = getTimerSeconds();
  startTimer();

  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, deviceOutput, inputLength * sizeof(DataType), hipMemcpyDeviceToHost);

  timeDeviceToHost = getTimerSeconds();

  printf("hipMemcpy (host to device) time (s): %f\nvecAdd kernel time (s): %f\ncudaMemcpy (device to host) time (s): %f\nTotal time (s): %f\n", timeHostToDevice, timeKernel, timeDeviceToHost, timeHostToDevice+timeKernel+timeDeviceToHost);

  //@@ Insert code below to compare the output with the reference
  int diffs = 0;

  // Loops through vectors to check that the addition was correct
  for (int i = 0; i < inputLength; ++i) {
    // Not correct
    if (abs(hostOutput[i] - resultRef[i]) > 0.00001) {
      printf("Elements %i differentiates, found %f instead of %f\n", i, hostOutput[i], resultRef[i]);
      diffs++;
    }
  }

  printf("Number of errors: %d/%d\n", diffs, inputLength);

  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  //@@ Free the CPU memory here
  free(hostInput1);
  free(hostInput2);
  free(hostOutput);
  free(resultRef);

  printf("Done.\n");

  return 0;
}
