#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>

#define DataType double
#define MAXIMUM_RANDOM_VALUE 1000
#define TOLERANCE 0.001 // For double
// #define TOLERANCE 1000 // For float

struct timeval startTime;

void startTimer() {
  // Used "Tutorial: Timing your Kernel - CPU Timer & nvprof" for reference
  gettimeofday(&startTime, NULL);
}

double getTimerSeconds() {
  // Used "Tutorial: Timing your Kernel - CPU Timer & nvprof" for reference
  struct timeval endTime;
  gettimeofday(&endTime, NULL);
  return ((double)endTime.tv_sec + (double)endTime.tv_usec * 1.e-6) - ((double)startTime.tv_sec + (double)startTime.tv_usec * 1.e-6);
}

// Compute C = A * B
__global__ void gemm(DataType *A, DataType *B, DataType *C, int numARows, int numAColumns, int numBRows, int numBColumns){
  //@@ Insert code to implement matrix multiplication here
  const int rowC = blockIdx.y;
  const int colC = blockIdx.x;

  const int rowA = rowC;
  const int colA = threadIdx.x;

  const int rowB = threadIdx.x;
  const int colB = colC;

  if (rowA < numARows && colA < numAColumns && colB < numBColumns) {
    atomicAdd(&C[colC + rowC * numBColumns], A[colA + rowA * numAColumns] * B[colB + rowB * numBColumns]);
  }
}

int main(int argc, char **argv) {
  DataType *unifiedA; // The A matrix
  DataType *unifiedB; // The B matrix
  DataType *unifiedC; // The output C matrix
  DataType *resultRef; // The reference result
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;
  int numCColumns;
  double timeTotal;
  int prefetching = 0;

  //@@ Insert code below to read in numARows, numAColumns, numBColumns from args
  if (argc != 4 && argc != 5) {
    printf("Incorrect number of arguments.\nUsage: ./lab4_ex3_unified.out <numARows> <numAColumns> <numBColumns> [<prefetching>]\n");
    return -1;
  }

  numARows = atoi(argv[1]);
  numAColumns = atoi(argv[2]);

  numBRows = numAColumns;
  numBColumns = atoi(argv[3]);

  numCRows = numARows;
  numCColumns = numBColumns;

  if (argc == 5) {
    prefetching = atoi(argv[4]);
  }

  printf("Input matrix dim (%d x %d) (%d x %d) (%d x %d)\n", numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
  printf("Prefetching: %i\n", prefetching);

  //@@ Insert code below to allocate memory here
  hipMallocManaged(&unifiedA, numARows * numAColumns * sizeof(DataType));
  hipMallocManaged(&unifiedB, numBRows * numBColumns * sizeof(DataType));
  hipMallocManaged(&unifiedC, numCRows * numCColumns * sizeof(DataType));
  resultRef = (DataType*) calloc(numCRows * numCColumns, numCRows * numCColumns * sizeof(DataType));

  // Prefetch unified memory to host
  if (prefetching == 1) {
    hipMemPrefetchAsync(unifiedA, numARows * numAColumns * sizeof(DataType), hipCpuDeviceId);
    hipMemPrefetchAsync(unifiedB, numBRows * numBColumns * sizeof(DataType), hipCpuDeviceId);
  }

  //@@ Insert code below to initialize hostA and hostB to random numbers, and create reference result in CPU
  srand(time(NULL));

  for (int i = 0; i < numARows * numAColumns; ++i) {
    unifiedA[i] = ((DataType)rand() / (DataType)RAND_MAX) * MAXIMUM_RANDOM_VALUE;
  }

  for (int i = 0; i < numBRows * numBColumns; ++i) {
    unifiedB[i] = ((DataType)rand() / (DataType)RAND_MAX) * MAXIMUM_RANDOM_VALUE;
  }

  // Traverse each column
  for (int y = 0; y < numCRows; ++y) {
    // Traverse each row
    for (int x = 0; x < numCColumns; ++x) {
      // Elements to multiply
      for (int i = 0; i < numAColumns; ++i) {
        resultRef[x + y * numCColumns] += unifiedA[i + y * numAColumns] * unifiedB[x + i * numBColumns];
      }
    }
  }

  // Prefetch unified memory to device
  if (prefetching == 1) {
    hipMemPrefetchAsync(unifiedA, numARows * numAColumns * sizeof(DataType), 0);
    hipMemPrefetchAsync(unifiedB, numBRows * numBColumns * sizeof(DataType), 0);
    hipMemPrefetchAsync(unifiedC, numCRows * numCColumns * sizeof(DataType), 0);
  }

  //@@ Initialize the grid and block dimensions here
  dim3 block(numAColumns); // One thread for each multiplication (1D)
  dim3 grid(numCColumns, numCRows); // One block for each element in the C matrix (2D)

  startTimer();

  //@@ Launch the GPU Kernel here
  gemm<<<grid, block>>>(unifiedA, unifiedB, unifiedC, numARows, numAColumns, numBRows, numBColumns);
  hipDeviceSynchronize();

  timeTotal = getTimerSeconds();

  // Prefetch unified memory to host
  if (prefetching == 1) {
    hipMemPrefetchAsync(unifiedC, numCRows * numCColumns * sizeof(DataType), hipCpuDeviceId);
  }

  //@@ Insert code below to compare the output with the reference
  int diffs = 0;

  for (int i = 0; i < numCRows * numCColumns; ++i) {
    // Not correct
    if (abs(unifiedC[i] - resultRef[i]) > TOLERANCE) {
      int x = i % numCColumns;
      int y = i / numCColumns;

      printf("Elements (%i, %i) differentiates, found %f instead of %f\n", x, y, unifiedC[i], resultRef[i]);
      diffs++;
    }
  }

  printf("Total time (s): %f\n", timeTotal);

  printf("Number of errors: %d/%d\n", diffs, numCRows * numCColumns);

  //@@ Free the Unified memory here
  hipFree(unifiedA);
  hipFree(unifiedB);
  hipFree(unifiedC);
  free(resultRef);

  printf("Done.\n");

  return 0;
}
