#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>

#define DataType double
#define THREADS_PER_BLOCK 64
#define MAXIMUM_RANDOM_VALUE 1000

struct timeval startTime;

//@@ Insert code to implement timer start
void startTimer() {
  // Used "Tutorial: Timing your Kernel - CPU Timer & nvprof" for reference
  gettimeofday(&startTime, NULL);
}

//@@ Insert code to implement timer stop
double getTimerSeconds() {
  // Used "Tutorial: Timing your Kernel - CPU Timer & nvprof" for reference
  struct timeval endTime;
  gettimeofday(&endTime, NULL);
  return ((double)endTime.tv_sec + (double)endTime.tv_usec * 1.e-6) - ((double)startTime.tv_sec + (double)startTime.tv_usec * 1.e-6);
}

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
  //@@ Insert code to implement vector addition here
  const int id = blockIdx.x * blockDim.x + threadIdx.x;

  if (id < len) {
    out[id] = in1[id] + in2[id];
  }
}

int main(int argc, char **argv) {
  int inputLength;
  int numStreams;
  int segmentSize; // S_seg
  DataType *hostInput1;
  DataType *hostInput2;
  DataType *hostOutput;
  DataType *resultRef;
  DataType *deviceInput1;
  DataType *deviceInput2;
  DataType *deviceOutput;

  //@@ Insert code below to read in inputLength from args
  if (argc != 4 || (argv[2] == "-" && argv[3] == "-")) {
    printf("Incorrect number of arguments.\nUsage (use - to autofill either segmentSize or numStreams): ./lab4_ex2.out <inputLength> <segmentSize> <numStreams>\n");
    return -1;
  }

  inputLength = atoi(argv[1]);

  // Automatically fill segmentSize
  if (strcmp(argv[2], "-") == 0) {
    numStreams = atoi(argv[3]);
    segmentSize = (inputLength + numStreams - 1) / numStreams;
  }
  // Automatically fill numStreams
  else if (strcmp(argv[3], "-") == 0) {
    segmentSize = atoi(argv[2]);
    numStreams = (inputLength + segmentSize - 1) / segmentSize;
  }
  // Both segmentSize and numStreams were specified
  else {
    segmentSize = atoi(argv[2]);
    numStreams = atoi(argv[3]);
  }

  printf("The input length is %i\nThe segment size is %i\nThe number of streams are %i\n", inputLength, segmentSize, numStreams);

  //@@ Insert code below to allocate Host memory for input and output
  hipHostAlloc(&hostInput1, inputLength * sizeof(DataType), hipHostMallocDefault);
  hipHostAlloc(&hostInput2, inputLength * sizeof(DataType), hipHostMallocDefault);
  hipHostAlloc(&hostOutput, inputLength * sizeof(DataType), hipHostMallocDefault);
  hipHostAlloc(&resultRef, inputLength * sizeof(DataType), hipHostMallocDefault);

  //@@ Insert code below to initialize hostInput1 and hostInput2 to random numbers, and create reference result in CPU
  srand(time(NULL));

  for (int i = 0; i < inputLength; ++i) {
    hostInput1[i] = ((float)rand() / (float)RAND_MAX) * MAXIMUM_RANDOM_VALUE;
    hostInput2[i] = ((float)rand() / (float)RAND_MAX) * MAXIMUM_RANDOM_VALUE;
    resultRef[i] = hostInput1[i] + hostInput2[i];
  }

  //@@ Insert code below to allocate GPU memory here
  hipMalloc(&deviceInput1, inputLength * sizeof(DataType));
  hipMalloc(&deviceInput2, inputLength * sizeof(DataType));
  hipMalloc(&deviceOutput, inputLength * sizeof(DataType));

  //@@ Initialize the 1D grid and block dimensions here
  dim3 block(THREADS_PER_BLOCK);
  dim3 grid((segmentSize + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);

  startTimer();

  // Create streams
  hipStream_t streams[numStreams];

  for (int i = 0; i < numStreams; ++i) {
    hipStreamCreate(&streams[i]);
  }

  int processedInputValues = 0; // Number of input values out of inputLength that has been processed so far

  while (processedInputValues < inputLength) {
    for (int i = 0; i < numStreams; ++i) {
      int offset = processedInputValues; // Start of current segment
      int valuesInSegment = segmentSize; // Number of values in this segment

      // Do not process more values than there was in the input
      if (offset + valuesInSegment >= inputLength) {
        valuesInSegment = inputLength - offset;
      }

      //@@ Insert code to below to Copy memory to the GPU here
      hipMemcpyAsync(&deviceInput1[offset], &hostInput1[offset], valuesInSegment * sizeof(DataType), hipMemcpyHostToDevice, streams[i]);
      hipMemcpyAsync(&deviceInput2[offset], &hostInput2[offset], valuesInSegment * sizeof(DataType), hipMemcpyHostToDevice, streams[i]);

      //@@ Launch the GPU Kernel here
      vecAdd<<<grid, block, 0, streams[i]>>>(&deviceInput1[offset], &deviceInput2[offset], &deviceOutput[offset], valuesInSegment);

      //@@ Copy the GPU memory back to the CPU here
      hipMemcpyAsync(&hostOutput[offset], &deviceOutput[offset], valuesInSegment * sizeof(DataType), hipMemcpyDeviceToHost, streams[i]);

      processedInputValues += valuesInSegment;

      // Already processed all values
      if (processedInputValues >= inputLength) {
        break;
      }
    }
  }

  hipDeviceSynchronize();

  // Destroy streams
  for (int i = 0; i < numStreams; ++i) {
    hipStreamDestroy(streams[i]);
  }

  printf("Total time (s): %f\n", getTimerSeconds());

  //@@ Insert code below to compare the output with the reference
  int diffs = 0;

  // Loops through vectors to check that the addition was correct
  for (int i = 0; i < inputLength; ++i) {
    // Not correct
    if (abs(hostOutput[i] - resultRef[i]) > 0.00001) {
      printf("Elements %i differentiates, found %f instead of %f\n", i, hostOutput[i], resultRef[i]);
      diffs++;
    }
  }

  printf("Number of errors: %d/%d\n", diffs, inputLength);

  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  //@@ Free the CPU memory here
  hipHostFree(hostInput1);
  hipHostFree(hostInput2);
  hipHostFree(hostOutput);
  hipHostFree(resultRef);

  printf("Done.\n");

  return 0;
}
